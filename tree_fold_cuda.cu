#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <set>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/execution_policy.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <unordered_map>
#include <fstream>
#include <string>
#include <map>
#include <cctype>
#include <sstream>
#include <cassert>
#include <cstdint>
#include <execution>
#include <functional>
#include <malloc.h>
#include <zstd.h>
#include "json.hpp"

using json = nlohmann::json;

// Error checking macro
#define CHECK_CUDA_ERROR(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

// Configuration parameters
#define MAX_ELEMENTS_PER_VECTOR 128
#define BLOCK_SIZE 256
#define TILE_SIZE_A 256  // Tile size for set A in tiled processing
#define TILE_SIZE_B 3072 // Tile size for set B in tiled processing
#define RESULTS_FLUSH_THRESHOLD 10000 // In-memory result limit before flushing to disk
#define CHUNK_SIZE (1024 * 4) // Number of items to load from a stream at a time

// --- Forward Declarations ---
typedef struct {
    int8_t* data;         // Flattened array of all elements
    int* offsets;      // Starting index for each vector/set
    int* sizes;        // Size of each vector/set
    int numItems;      // Number of vectors/sets
    int totalElements; // Total number of elements
    int8_t* deviceBuffer; // Reusable device buffer for operations
    int bufferSize;    // Size of the device buffer
} CudaSet;
struct LevelItem;
struct ProcessResult;
ProcessResult processPair_inMemory(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose);
void processLargePair(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose, 
                     std::unordered_map<size_t, std::vector<int>>& uniqueResults,
                     std::function<void()> flushCallback = nullptr);
ProcessResult processPair(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose, bool allowStreaming);
LevelItem processStreamedPair(LevelItem& itemA, LevelItem& itemB, int threshold, int level, bool verbose);

// Absolute value functor for Thrust
struct AbsoluteFunctor {
    __host__ __device__
    int operator()(const int x) const {
        return x < 0 ? -x : x;
    }
};

// Function to read Witness JSON and generate test sets
std::vector<std::vector<std::vector<int>>> generateWitnessSetsFromJSON(const std::string& filename) {
    // Read JSON file
    std::ifstream file(filename);
    if (!file.is_open()) {
        printf("Error: Could not open file: %s\n", filename.c_str());
        return {};
    }
    
    // Parse JSON
    json j;
    try {
        file >> j;
    } catch (const json::exception& e) {
        printf("Error parsing JSON: %s\n", e.what());
        return {};
    }
    
    // Validate JSON structure
    if (!j.contains("clauses") || !j["clauses"].is_array()) {
        printf("Error: JSON must contain 'clauses' array\n");
        return {};
    }
    
    std::vector<std::vector<std::vector<int>>> testSets;
    
    // Process each clause
    for (const auto& clause : j["clauses"]) {
        if (!clause.contains("name") || !clause.contains("assignments")) {
            printf("Warning: Skipping clause without 'name' or 'assignments'\n");
            continue;
        }
        
        std::string clauseName = clause["name"];
        const auto& assignments = clause["assignments"];
        
        if (!assignments.is_array()) {
            printf("Warning: Skipping clause '%s' - 'assignments' is not an array\n", clauseName.c_str());
            continue;
        }
        
        std::vector<std::vector<int>> clauseSet;
        
        // Process each assignment
        for (const auto& assignment : assignments) {
            if (!assignment.is_array()) {
                printf("Warning: Skipping non-array assignment in clause '%s'\n", clauseName.c_str());
                continue;
            }
            
            std::vector<int> assignmentVec;
            for (const auto& value : assignment) {
                if (value.is_number()) {
                    assignmentVec.push_back(value.get<int>());
                } else {
                    printf("Warning: Skipping non-numeric value in assignment\n");
                }
            }
            
            if (!assignmentVec.empty()) {
                clauseSet.push_back(assignmentVec);
            }
        }
        
        printf("  Clause '%s': %zu assignments\n", clauseName.c_str(), clauseSet.size());
        testSets.push_back(clauseSet);
    }
    
    printf("\nGenerated %zu test sets from Witness JSON\n\n", testSets.size());
    
    // Print verification of the testSets
    printf("=== Verification of Generated Test Sets ===\n");
    for (size_t i = 0; i < testSets.size(); i++) {
        printf("Test Set %zu:\n", i);
        printf("{\n");
        for (size_t j = 0; j < testSets[i].size(); j++) {
            printf("  {");
            for (size_t k = 0; k < testSets[i][j].size(); k++) {
                printf("%d", testSets[i][j][k]);
                if (k < testSets[i][j].size() - 1) {
                    printf(",");
                }
            }
            printf("}");
            if (j < testSets[i].size() - 1) {
                printf(",");
            }
            printf("\n");
        }
        printf("}\n\n");
    }
    
    return testSets;
}

//-------------------------------------------------------------------------
// Host-side data structures
//-------------------------------------------------------------------------
typedef struct {
    std::vector<std::vector<int>> vectors;  // Original vectors 
} HostSet;

// Result buffer for parallel combination processing
typedef struct {
    int* data;         // Buffer for all potential results
    int* validFlags;   // Flags indicating if each combination is valid
    int* sizes;        // Size of each result set
    int maxResultSize; // Maximum possible size of a result
    int numCombinations; // Total number of combinations
} CombinationResultBuffer;

// Result struct to handle in-memory or streamed results
struct ProcessResult {
    CudaSet set;
    std::string streamPath; // Path to file if results are streamed
    int fromIdA = -1;
    int fromIdB = -1;
    size_t numResultItems = 0;
};

// Allocate memory for a CUDA set with additional buffer space
CudaSet allocateCudaSet(int numItems, int totalElements, int bufferSize = 0) {
    CudaSet set;
    set.numItems = numItems;
    set.totalElements = totalElements;
    
    CHECK_CUDA_ERROR(hipMalloc(&set.data, totalElements * sizeof(int8_t)));
    CHECK_CUDA_ERROR(hipMalloc(&set.offsets, numItems * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&set.sizes, numItems * sizeof(int)));
    
    // Allocate device buffer if size is specified
    if (bufferSize > 0) {
        CHECK_CUDA_ERROR(hipMalloc(&set.deviceBuffer, bufferSize * sizeof(int8_t)));
        set.bufferSize = bufferSize;
    } else {
        set.deviceBuffer = nullptr;
        set.bufferSize = 0;
    }
    
    return set;
}

// Free memory for a CUDA set
void freeCudaSet(CudaSet* set) {
    if (set->data) hipFree(set->data);
    if (set->offsets) hipFree(set->offsets);
    if (set->sizes) hipFree(set->sizes);
    if (set->deviceBuffer) hipFree(set->deviceBuffer);
    set->numItems = 0;
    set->totalElements = 0;
    set->bufferSize = 0;
    set->data = nullptr;
    set->offsets = nullptr;
    set->sizes = nullptr;
    set->deviceBuffer = nullptr;
}

// Allocate result buffer for parallel combination processing
CombinationResultBuffer allocateCombinationResultBuffer(int numItemsA, int numItemsB, int maxElementsPerVector) {
    CombinationResultBuffer buffer;
    buffer.numCombinations = numItemsA * numItemsB;
    buffer.maxResultSize = 2 * maxElementsPerVector; // Worst case: all elements from both vectors
    
    CHECK_CUDA_ERROR(hipMalloc(&buffer.data, buffer.numCombinations * buffer.maxResultSize * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&buffer.validFlags, buffer.numCombinations * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&buffer.sizes, buffer.numCombinations * sizeof(int)));
    
    // Initialize all valid flags to 0 (invalid)
    CHECK_CUDA_ERROR(hipMemset(buffer.validFlags, 0, buffer.numCombinations * sizeof(int)));
    
    return buffer;
}

// Free result buffer
void freeCombinationResultBuffer(CombinationResultBuffer* buffer) {
    if (buffer->data) hipFree(buffer->data);
    if (buffer->validFlags) hipFree(buffer->validFlags);
    if (buffer->sizes) hipFree(buffer->sizes);
    buffer->data = nullptr;
    buffer->validFlags = nullptr;
    buffer->sizes = nullptr;
}

// Host to device copy for a set (optimized to use pinned memory for larger transfers)
void copyHostToDevice(const HostSet& hostSet, CudaSet* cudaSet) {
    int numItems = hostSet.vectors.size();
    
    // Prepare host side arrays
    std::vector<int> hostIntData;
    std::vector<int> hostOffsets(numItems);
    std::vector<int> hostSizes(numItems);
    
    int currentOffset = 0;
    for (int i = 0; i < numItems; i++) {
        hostOffsets[i] = currentOffset;
        hostSizes[i] = hostSet.vectors[i].size();
        
        for (int j = 0; j < hostSet.vectors[i].size(); j++) {
            hostIntData.push_back(hostSet.vectors[i][j]);
        }
        
        currentOffset += hostSet.vectors[i].size();
    }
    
    // Convert to int8_t for device storage
    std::vector<int8_t> hostData(hostIntData.size());
    for (size_t i = 0; i < hostIntData.size(); ++i) {
        assert(hostIntData[i] >= INT8_MIN && hostIntData[i] <= INT8_MAX && "Input data exceeds int8_t range!");
        hostData[i] = static_cast<int8_t>(hostIntData[i]);
    }

    // Use pinned memory for large transfers
    int totalElements = hostData.size();
    int8_t* pinnedData = nullptr;
    int* pinnedOffsets = nullptr;
    int* pinnedSizes = nullptr;
    
    if (totalElements > 1024) {
        CHECK_CUDA_ERROR(hipHostMalloc((void**)&pinnedData, totalElements * sizeof(int8_t)));
        CHECK_CUDA_ERROR(hipHostMalloc(&pinnedOffsets, numItems * sizeof(int)));
        CHECK_CUDA_ERROR(hipHostMalloc(&pinnedSizes, numItems * sizeof(int)));
        
        memcpy(pinnedData, hostData.data(), totalElements * sizeof(int8_t));
        memcpy(pinnedOffsets, hostOffsets.data(), numItems * sizeof(int));
        memcpy(pinnedSizes, hostSizes.data(), numItems * sizeof(int));
    }
    
    // Allocate device memory
    *cudaSet = allocateCudaSet(numItems, totalElements, totalElements * 2);
    
    // Copy data to device
    if (totalElements > 1024) {
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->data, pinnedData, totalElements * sizeof(int8_t), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->offsets, pinnedOffsets, numItems * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->sizes, pinnedSizes, numItems * sizeof(int), hipMemcpyHostToDevice));
        
        hipHostFree(pinnedData);
        hipHostFree(pinnedOffsets);
        hipHostFree(pinnedSizes);
    } else {
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->data, hostData.data(), totalElements * sizeof(int8_t), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->offsets, hostOffsets.data(), numItems * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA_ERROR(hipMemcpy(cudaSet->sizes, hostSizes.data(), numItems * sizeof(int), hipMemcpyHostToDevice));
    }
}

// Device to host copy (optimized with streams for larger data)
HostSet copyDeviceToHost(const CudaSet& cudaSet) {
    HostSet hostSet;
    
    // Copy offsets and sizes
    std::vector<int> hostOffsets(cudaSet.numItems);
    std::vector<int> hostSizes(cudaSet.numItems);
    
    CHECK_CUDA_ERROR(hipMemcpy(hostOffsets.data(), cudaSet.offsets, cudaSet.numItems * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), cudaSet.sizes, cudaSet.numItems * sizeof(int), hipMemcpyDeviceToHost));
    
    // For large data, use async transfers with streams
    std::vector<int8_t> hostData8(cudaSet.totalElements);
    
    if (cudaSet.totalElements > 1024) {
        hipStream_t stream;
        CHECK_CUDA_ERROR(hipStreamCreate(&stream));
        
        int8_t* pinnedData;
        CHECK_CUDA_ERROR(hipHostMalloc((void**)&pinnedData, cudaSet.totalElements * sizeof(int8_t)));
        
        CHECK_CUDA_ERROR(hipMemcpyAsync(pinnedData, cudaSet.data, cudaSet.totalElements * sizeof(int8_t), 
                                       hipMemcpyDeviceToHost, stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
        
        memcpy(hostData8.data(), pinnedData, cudaSet.totalElements * sizeof(int8_t));
        
        hipHostFree(pinnedData);
        hipStreamDestroy(stream);
    } else {
        CHECK_CUDA_ERROR(hipMemcpy(hostData8.data(), cudaSet.data, cudaSet.totalElements * sizeof(int8_t), 
                                  hipMemcpyDeviceToHost));
    }
    
    // Reconstruct vectors
    hostSet.vectors.resize(cudaSet.numItems);
    
    // Convert back to int
    std::vector<int> hostData(cudaSet.totalElements);
    for (size_t i = 0; i < hostData8.size(); ++i) {
        hostData[i] = hostData8[i];
    }

    for (int i = 0; i < cudaSet.numItems; i++) {
        int offset = hostOffsets[i];
        int size = hostSizes[i];
        
        hostSet.vectors[i].resize(size);
        for (int j = 0; j < size; j++) {
            hostSet.vectors[i][j] = hostData[offset + j];
        }
    }
    
    return hostSet;
}

// Helper function to create a test set
HostSet createTestSet(const std::vector<std::vector<int>>& vectors) {
    HostSet set;
    set.vectors = vectors;
    return set;
}

//-------------------------------------------------------------------------
// CUDA Kernels and Device Functions
//-------------------------------------------------------------------------

// Device function to check if an element is in a set
__device__ bool deviceContains(const int* array, int size, int value) {
    for (int i = 0; i < size; i++) {
        if (array[i] == value) {
            return true;
        }
    }
    return false;
}

// Kernel to convert vector elements to unique elements (for Level 1 carry-over)
__global__ void convertToUniqueKernel(
    int8_t* inputData, int* inputOffsets, int* inputSizes, int numItems,
    int8_t* outputData, int* outputOffsets, int* outputSizes, int maxOutputSize
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= numItems) {
        return;
    }
    
    int inputOffset = inputOffsets[idx];
    int inputSize = inputSizes[idx];
    int outputOffset = outputOffsets[idx];
    
    // Local working memory for unique elements
    int localSet[MAX_ELEMENTS_PER_VECTOR];
    int localSetSize = 0;
    
    // Get unique elements
    for (int i = 0; i < inputSize; i++) {
        int val = inputData[inputOffset + i];
        if (!deviceContains(localSet, localSetSize, val)) {
            localSet[localSetSize++] = val;
        }
    }
    
    // Copy result to output
    outputSizes[idx] = localSetSize;
    for (int i = 0; i < localSetSize; i++) {
        outputData[outputOffset + i] = localSet[i];
    }
}

// Kernel that processes all combinations with built-in batching
__global__ void processAllCombinationsKernel(
    int8_t* dataA, int* offsetsA, int* sizesA, int numItemsA,
    int8_t* dataB, int* offsetsB, int* sizesB, int numItemsB,
    int threshold, int level,
    int* resultData, int* resultSizes, int* validFlags, int maxResultSize,
    int combinationsPerThread
) {
    // Calculate global thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread processes multiple combinations using grid-stride loop
    for (int i = 0; i < combinationsPerThread; i++) {
        // Calculate combination index for this thread and iteration
        int combinationIdx = threadId * combinationsPerThread + i;
        
        // Check if this combination index is valid
        if (combinationIdx >= numItemsA * numItemsB) {
            return;
        }
        
        // Calculate setA and setB indices from the combination index
        int idxA = combinationIdx / numItemsB;
        int idxB = combinationIdx % numItemsB;
        
        // Get vectors from set A and set B
        int offsetA = offsetsA[idxA];
        int sizeA = sizesA[idxA];
        int offsetB = offsetsB[idxB];
        int sizeB = sizesB[idxB];
        
        // Local working memory for unique elements
        int localSet[MAX_ELEMENTS_PER_VECTOR * 2];
        int localSetSize = 0;
        
        // Merge vectors, keeping only unique elements
        for (int j = 0; j < sizeA; j++) {
            int val = dataA[offsetA + j];
            if (!deviceContains(localSet, localSetSize, val)) {
                localSet[localSetSize++] = val;
            }
        }
        
        for (int j = 0; j < sizeB; j++) {
            int val = dataB[offsetB + j];
            if (!deviceContains(localSet, localSetSize, val)) {
                localSet[localSetSize++] = val;
            }
        }
        
        // Check threshold condition
        bool isValid = (threshold == 0 || localSetSize <= threshold);
        
        // If valid, copy result to output buffer
        if (isValid) {
            validFlags[combinationIdx] = 1;
            resultSizes[combinationIdx] = localSetSize;
            
            int resultOffset = combinationIdx * maxResultSize;
            for (int j = 0; j < localSetSize; j++) {
                resultData[resultOffset + j] = localSet[j];
            }
        } else {
            validFlags[combinationIdx] = 0;
            resultSizes[combinationIdx] = localSetSize; // Store size for debugging
        }
    }
}

//-------------------------------------------------------------------------
// Core processing functions
//-------------------------------------------------------------------------

// Represents an item in a processing level of the tree fold
struct LevelItem {
    CudaSet set;
    std::string streamPath;
    int numItems;
    int id;
    bool needsCleanup; // True if this is an intermediate result that should be freed/deleted

    bool isStreamed() const { return !streamPath.empty(); }
};

// Global counter for unique item IDs
static int levelItemCounter = 0;

// Helper to get the first vector from a CudaSet for threshold calculation
std::vector<int> getFirstVectorFromCudaSet(const CudaSet& set) {
    if (set.numItems == 0) return {};
    int size;
    CHECK_CUDA_ERROR(hipMemcpy(&size, set.sizes, sizeof(int), hipMemcpyDeviceToHost));
    
    std::vector<int8_t> h_firstVector8(size);
    int offset = 0; // First vector is always at offset 0
    CHECK_CUDA_ERROR(hipMemcpy(h_firstVector8.data(), set.data + offset, size * sizeof(int8_t), hipMemcpyDeviceToHost));
    
    std::vector<int> firstVector(size);
    for(int i = 0; i < size; ++i) firstVector[i] = h_firstVector8[i];
    return firstVector;
}

// Helper to get the first vector from a streamed file
std::vector<int> getFirstVectorFromStream(const std::string& filePath) {
    FILE* inFile = fopen(filePath.c_str(), "rb");
    if (!inFile) return {};

    int vecSize = 0;
    size_t elementsRead = fread(&vecSize, sizeof(int), 1, inFile);
    if (elementsRead == 0) {
        fclose(inFile);
        return {};
    }

    std::vector<int> firstVec(vecSize);
    fread(firstVec.data(), sizeof(int), vecSize, inFile);
    fclose(inFile);
    return firstVec;
}

// Modified threshold computation to handle streamed and in-memory sets
int computeThreshold(const LevelItem& itemA, const LevelItem& itemB) {
    if (itemA.numItems == 0 || itemB.numItems == 0) return 0;

    // Get the first vector from item A
    std::vector<int> firstVectorA = itemA.isStreamed() ? 
        getFirstVectorFromStream(itemA.streamPath) : 
        getFirstVectorFromCudaSet(itemA.set);

    // Get the first vector from item B
    std::vector<int> firstVectorB = itemB.isStreamed() ? 
        getFirstVectorFromStream(itemB.streamPath) : 
        getFirstVectorFromCudaSet(itemB.set);

    if (firstVectorA.empty() || firstVectorB.empty()) return 0;
    
    // The rest of the logic is the same: find unique absolute values
    std::set<int> uniqueAbsValues;
    for (int value : firstVectorA) uniqueAbsValues.insert(abs(value));
    for (int value : firstVectorB) uniqueAbsValues.insert(abs(value));
        
    return uniqueAbsValues.size();
}

// Helper function to extract a subset from a CudaSet
CudaSet extractSubset(const CudaSet& set, int startIndex, int count, bool verbose) {
    if (count <= 0) {
        // Return empty set
        CudaSet emptySet;
        emptySet.numItems = 0;
        emptySet.totalElements = 0;
        emptySet.data = nullptr;
        emptySet.offsets = nullptr;
        emptySet.sizes = nullptr;
        emptySet.deviceBuffer = nullptr;
        emptySet.bufferSize = 0;
        return emptySet;
    }
    
    // Copy size and offset information for the slice
    std::vector<int> hostSizes(count);
    std::vector<int> hostOffsets(count);
    
    CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), set.sizes + startIndex, 
                              count * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(hostOffsets.data(), set.offsets + startIndex, 
                              count * sizeof(int), hipMemcpyDeviceToHost));
    
    // Calculate total elements in the subset
    int totalElements = 0;
    for (int i = 0; i < count; i++) {
        totalElements += hostSizes[i];
    }
    
    // Allocate memory for the subset
    CudaSet subSet = allocateCudaSet(count, totalElements);
    
    // Copy offset and size information
    std::vector<int> newOffsets(count);
    int currentOffset = 0;
    for (int i = 0; i < count; i++) {
        newOffsets[i] = currentOffset;
        currentOffset += hostSizes[i];
    }
    
    CHECK_CUDA_ERROR(hipMemcpy(subSet.sizes, hostSizes.data(), 
                              count * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(subSet.offsets, newOffsets.data(), 
                              count * sizeof(int), hipMemcpyHostToDevice));
    
    // Copy data elements for each vector
    for (int i = 0; i < count; i++) {
        int srcOffset = hostOffsets[i];
        int dstOffset = newOffsets[i];
        int size = hostSizes[i];
        
        CHECK_CUDA_ERROR(hipMemcpy(subSet.data + dstOffset, set.data + srcOffset, 
                                  size * sizeof(int8_t), hipMemcpyDeviceToDevice));
    }
    
    return subSet;
}

// An internal version of processPair that is guaranteed to run on the GPU without triggering another streaming operation.
// It also contains the full deduplication logic.
ProcessResult processPair_inMemory(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose) {
    long long totalCombinations = (long long)setA.numItems * (long long)setB.numItems;
    
    // Calculate buffer size needed
    int maxResultsPerThread = 4;
    int threadsNeeded = (totalCombinations + maxResultsPerThread - 1) / maxResultsPerThread;
    
    // Determine thread block configuration
    int threadsPerBlock = 256;
    int blocksNeeded = (threadsNeeded + threadsPerBlock - 1) / threadsPerBlock;
    
    // Limit blocks to avoid excessive memory usage
    const int MAX_BLOCKS = 16384;
    if (blocksNeeded > MAX_BLOCKS) {
        blocksNeeded = MAX_BLOCKS;
        maxResultsPerThread = (totalCombinations + (blocksNeeded * threadsPerBlock) - 1) / (blocksNeeded * threadsPerBlock);
    }
    
    // Allocate result buffer
    CombinationResultBuffer resultBuffer = allocateCombinationResultBuffer(setA.numItems, setB.numItems, MAX_ELEMENTS_PER_VECTOR);
    
    // Launch kernel
    processAllCombinationsKernel<<<blocksNeeded, threadsPerBlock>>>(
        setA.data, setA.offsets, setA.sizes, setA.numItems,
        setB.data, setB.offsets, setB.sizes, setB.numItems,
        threshold, level,
        resultBuffer.data, resultBuffer.sizes, resultBuffer.validFlags, resultBuffer.maxResultSize,
        maxResultsPerThread
    );
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Count valid combinations
    std::vector<int> hostValidFlags(resultBuffer.numCombinations);
    CHECK_CUDA_ERROR(hipMemcpy(hostValidFlags.data(), resultBuffer.validFlags, 
                              resultBuffer.numCombinations * sizeof(int), hipMemcpyDeviceToHost));
    
    int validCount = 0;
    for (int i = 0; i < resultBuffer.numCombinations; i++) {
        if (hostValidFlags[i]) validCount++;
    }

    if (verbose) {
        printf("    Found %d valid combinations out of %lld total\n", validCount, totalCombinations);
    }

    std::vector<std::vector<int>> validCombinations;
	if (validCount > 0) {
        if (verbose) {
		    printf("    Copying result data for %d valid combinations...\n", validCount);
		}

        std::vector<int> hostSizes(resultBuffer.numCombinations);
        CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), resultBuffer.sizes, 
                                  resultBuffer.numCombinations * sizeof(int), hipMemcpyDeviceToHost));
		
		std::vector<int> hostResultData(resultBuffer.numCombinations * resultBuffer.maxResultSize);
		CHECK_CUDA_ERROR(hipMemcpy(hostResultData.data(), resultBuffer.data,
		                          resultBuffer.numCombinations * resultBuffer.maxResultSize * sizeof(int),
		                          hipMemcpyDeviceToHost));
		
        // Progress reporting variables
		int reportInterval = validCount > 1000 ? validCount / 10 : validCount;
		int lastReportedCount = 0;
		int collectedCount = 0;

		for (int i = 0; i < resultBuffer.numCombinations; i++) {
		    if (hostValidFlags[i]) {
		        int size = hostSizes[i];
		        std::vector<int> combination(size);
		        int offset = i * resultBuffer.maxResultSize;
		        for (int j = 0; j < size; j++) {
		            combination[j] = hostResultData[offset + j];
		        }
		        validCombinations.push_back(combination);
                collectedCount++;
                
                // Progress reporting for large result sets
		        if (verbose && validCount > 1000 && collectedCount - lastReportedCount >= reportInterval) {
		            printf("    Collected %d of %d valid combinations (%.1f%%)\n", 
		                   collectedCount, validCount, 100.0 * collectedCount / validCount);
		            lastReportedCount = collectedCount;
		        }
		    }
		}

        if (verbose && validCount > 1000) {
		    printf("    Collection complete: %d combinations collected\n", collectedCount);
		}
	}

	freeCombinationResultBuffer(&resultBuffer);

    // Remove duplicates
    if (validCombinations.size() > 1) {
        if (verbose) {
            printf("    Deduplicating %zu combinations...\n", validCombinations.size());
        }
        for (auto& combination : validCombinations) {
            std::sort(combination.begin(), combination.end());
        }
        std::sort(std::execution::par, validCombinations.begin(), validCombinations.end());
        validCombinations.erase(std::unique(validCombinations.begin(), validCombinations.end()), validCombinations.end());
        if (verbose) {
            printf("    Deduplication complete: %zu unique combinations.\n", validCombinations.size());
        }
    }

    // Create result set
    HostSet resultHostSet;
    resultHostSet.vectors = validCombinations;
    
    CudaSet resultCudaSet;
    copyHostToDevice(resultHostSet, &resultCudaSet);
    
    return {resultCudaSet, ""};
}

ProcessResult processPair(const CudaSet& setA, const CudaSet& setB, int threshold, int level, bool verbose, bool allowStreaming = true) {
    int numItemsA = setA.numItems;
    int numItemsB = setB.numItems;
    
    if (verbose) {
        printf("  Processing pair at level %d: Set A (%d items) + Set B (%d items), threshold = %d\n", 
               level, numItemsA, numItemsB, threshold);
    }
    
    // Empty result for empty inputs
    if (numItemsA == 0 || numItemsB == 0) {
        CudaSet emptySet = {nullptr, nullptr, nullptr, 0, 0, nullptr, 0};
        return {emptySet, ""};
    }

    // For now, use in-memory processing for all cases
    if (verbose) {
        printf("    Using in-memory GPU processing.\n");
    }
    return processPair_inMemory(setA, setB, threshold, level, verbose);
}

// Special handling for converting a set to unique elements (for level 1 carry-over)
CudaSet convertSetToUnique(const CudaSet& set, bool verbose) {
    int numItems = set.numItems;
    
    // Allocate host vectors 
    std::vector<int> hostOffsets(numItems);
    std::vector<int> hostSizes(numItems);
    
    CHECK_CUDA_ERROR(hipMemcpy(hostSizes.data(), set.sizes, numItems * sizeof(int), hipMemcpyDeviceToHost));
    
    // Calculate max possible size for outputs
    int totalOutputSize = 0;
    for (int i = 0; i < numItems; i++) {
        totalOutputSize += hostSizes[i]; // Worst case: all elements are unique
    }
    
    // Create output arrays
    int8_t* d_outputData = nullptr;
    int* d_outputOffsets = nullptr;
    int* d_outputSizes = nullptr;
    
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_outputData, totalOutputSize * sizeof(int8_t)));
    CHECK_CUDA_ERROR(hipMalloc(&d_outputOffsets, numItems * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_outputSizes, numItems * sizeof(int)));
    
    // Calculate output offsets (equivalent to the input offsets)
    CHECK_CUDA_ERROR(hipMemcpy(d_outputOffsets, set.offsets, numItems * sizeof(int), hipMemcpyDeviceToDevice));
    
    // Launch parallel kernel
    int threadsPerBlock = 256;
    int blocks = (numItems + threadsPerBlock - 1) / threadsPerBlock;
    
    convertToUniqueKernel<<<blocks, threadsPerBlock>>>(
        set.data, set.offsets, set.sizes, numItems,
        d_outputData, d_outputOffsets, d_outputSizes, MAX_ELEMENTS_PER_VECTOR
    );
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    // Create result set
    CudaSet resultSet;
    resultSet.data = d_outputData;
    resultSet.offsets = d_outputOffsets;
    resultSet.sizes = d_outputSizes;
    resultSet.numItems = numItems;
    resultSet.totalElements = totalOutputSize;
    resultSet.deviceBuffer = nullptr;
    resultSet.bufferSize = 0;
    
    if (verbose) {
        printf("  Converting carried-over set for level 2\n");
        printf("  Carried over the last set with %d items\n", numItems);
    }
    
    return resultSet;
}

// Tree fold operations (simplified version without streaming)
LevelItem treeFoldOperations(const std::vector<CudaSet>& sets, bool verbose) {
    if (sets.empty()) {
        return { {nullptr, nullptr, nullptr, 0, 0, nullptr, 0}, "", 0, -1, false };
    }

    // Initialize the first level with LevelItems
    std::vector<LevelItem> currentLevel;
    for (const auto& s : sets) {
        currentLevel.push_back({s, "", s.numItems, levelItemCounter++, false});
    }

    if (currentLevel.size() == 1) {
        return currentLevel[0];
    }
    
    if (verbose) {
        printf("Starting tree-fold operations with %zu sets\n", sets.size());
        for (const auto& item : currentLevel) {
            printf("  Set %d: %d items\n", item.id, item.numItems);
        }
    }
    
    int level = 0;
    while (currentLevel.size() > 1) {
        level++;
        if (verbose) {
            printf("\nProcessing Level %d with %zu sets\n", level, currentLevel.size());
        }
        
        std::vector<LevelItem> nextLevel;
        
        // Process pairs
        for (size_t i = 0; i < currentLevel.size() - 1; i += 2) {
            LevelItem& itemA = currentLevel[i];
            LevelItem& itemB = currentLevel[i + 1];
            
            int threshold = computeThreshold(itemA, itemB);
            
            if (verbose) {
                printf("  --> Processing pair: Set %d (%d items) + Set %d (%d items) with threshold %d\n", 
                       itemA.id, itemA.numItems, itemB.id, itemB.numItems, threshold);
            }

            ProcessResult res = processPair(itemA.set, itemB.set, threshold, level, verbose, false);
            LevelItem resultItem = { res.set, res.streamPath, res.set.numItems, levelItemCounter++, true };
            nextLevel.push_back(resultItem);
        }
        
        // Handle odd set by carrying it over
        if (currentLevel.size() % 2 == 1) {
            LevelItem& carriedItem = currentLevel.back();
            if (verbose) {
                printf("  --> Carrying over odd set %d (%d items) to next level\n", 
                       carriedItem.id, carriedItem.numItems);
            }

            // For level 1, convert the carried-over set to unique elements
            if (level == 1) {
               CudaSet convertedSet = convertSetToUnique(carriedItem.set, verbose);
               nextLevel.push_back({convertedSet, "", convertedSet.numItems, levelItemCounter++, true});
            } else {
               carriedItem.needsCleanup = false;
               nextLevel.push_back(carriedItem);
            }
        }
        
        // Clean up resources from the completed level
        for(const auto& item : currentLevel) {
            if(item.needsCleanup) {
                freeCudaSet(&const_cast<CudaSet&>(item.set));
            }
        }
        
        currentLevel = nextLevel;
    }
    
    return currentLevel[0];
}

// Kernel to filter out negative values and sort
__global__ void filterAndSortKernel(int8_t* data, int* offsets, int* sizes, int numVectors, int maxLen) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numVectors) return;
    
    int offset = offsets[idx];
    int originalSize = sizes[idx];
    
    // Step 1: Filter out negatives
    int newSize = 0;
    for (int i = 0; i < originalSize; i++) {
        int val = data[offset + i];
        if (val >= 0) {
            // Keep only non-negative values
            data[offset + newSize] = val;
            newSize++;
        }
    }
    
    // Update size
    sizes[idx] = newSize;
    
    // Step 2: Sort (simple insertion sort)
    for (int i = 1; i < newSize; i++) {
        int key = data[offset + i];
        int j = i - 1;
        
        while (j >= 0 && data[offset + j] > key) {
            data[offset + j + 1] = data[offset + j];
            j--;
        }
        
        data[offset + j + 1] = key;
    }
}

// Function to post-process on GPU then complete ordering on CPU
std::vector<std::vector<int>> gpuPostProcess(const CudaSet& resultSet, bool verbose) {
    // Step 1: Run GPU kernel to filter and sort all vectors internally
    int threadsPerBlock = 256;
    int blocks = (resultSet.numItems + threadsPerBlock - 1) / threadsPerBlock;
    
    if (verbose) {
        printf("Running GPU post-processing on %d vectors\n", resultSet.numItems);
    }
    
    filterAndSortKernel<<<blocks, threadsPerBlock>>>(
        resultSet.data, resultSet.offsets, resultSet.sizes, 
        resultSet.numItems, MAX_ELEMENTS_PER_VECTOR);
    
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    
    if (verbose) {
        printf("GPU internal sorting complete, transferring to host for final sorting\n");
    }
    
    // Step 2: Process in batches to avoid memory issues
    const int BATCH_SIZE = 100000;
    int totalVectors = resultSet.numItems;
    int batches = (totalVectors + BATCH_SIZE - 1) / BATCH_SIZE;
    
    std::vector<std::vector<int>> processedResults;
    processedResults.reserve(std::min(totalVectors, 10000000)); // Reserve reasonable amount
    
    for (int batch = 0; batch < batches; batch++) {
        int start = batch * BATCH_SIZE;
        int end = std::min(start + BATCH_SIZE, totalVectors);
        
        if (verbose) {
            printf("Processing batch %d/%d (vectors %d to %d)\n", batch+1, batches, start, end-1);
        }
        
        // Extract subset of the CudaSet
        CudaSet batchSet = extractSubset(resultSet, start, end - start, false);
        
        // Process this batch - already filtered and sorted internally by GPU
        HostSet hostBatch = copyDeviceToHost(batchSet);
        
        // Add to results
        for (const auto& vector : hostBatch.vectors) {
            processedResults.push_back(vector);
        }
        
        // Free batch resources
        freeCudaSet(&batchSet);
        
        // Sort intermediate results if getting too large
        if (processedResults.size() > 1000000) {
            if (verbose) {
                printf("  Performing intermediate sort of %zu results\n", processedResults.size());
            }
            std::sort(processedResults.begin(), processedResults.end());
        }
    }
    
    // Final lexicographical sorting of all vectors
    if (verbose) {
        printf("Performing final lexicographical sort of %zu vectors\n", processedResults.size());
    }
    std::sort(processedResults.begin(), processedResults.end());
    
    return processedResults;
}

// Run test cases with Witness JSON
void runWitnessTestCases(const std::string& filename) {
    std::vector<std::vector<std::vector<int>>> testSets = 
        generateWitnessSetsFromJSON(filename);
    
    if (testSets.empty()) {
        printf("No test sets generated. Exiting.\n");
        return;
    }
    
    // Show input sets
    for (size_t i = 0; i < testSets.size(); i++) {
        printf("  Set %zu: [", i + 1);
        for (size_t j = 0; j < testSets[i].size() && j < 2; j++) {
            printf("[");
            for (size_t k = 0; k < testSets[i][j].size(); k++) {
                printf("%d", testSets[i][j][k]);
                if (k < testSets[i][j].size() - 1) printf(", ");
            }
            printf("]");
            if (j < testSets[i].size() - 1) printf(", ");
        }
        if (testSets[i].size() > 2) printf("...");
        printf("] (%zu items)\n", testSets[i].size());
    }
    
    // Create host sets
    std::vector<HostSet> hostSets;
    for (const auto& vectors : testSets) {
        hostSets.push_back(createTestSet(vectors));
    }
    
    // Convert host sets to CUDA sets
    std::vector<CudaSet> cudaSets;
    for (const auto& hostSet : hostSets) {
        CudaSet cudaSet;
        copyHostToDevice(hostSet, &cudaSet);
        cudaSets.push_back(cudaSet);
    }
    
    // Record start time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    // Run tree-fold operations
    LevelItem finalResult = treeFoldOperations(cudaSets, true);
    
    // Record end time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    printf("\nTree-fold completed in %.2f ms. Total items: %d\n", milliseconds, finalResult.numItems);
    
    std::vector<std::vector<int>> finalVectors;

    if (finalResult.numItems > 0) {
        printf("Final result is in memory (%d items). Post-processing on GPU...\n", finalResult.numItems);
        // Process the results from GPU memory
        finalVectors = gpuPostProcess(finalResult.set, true);
        freeCudaSet(&const_cast<CudaSet&>(finalResult.set));
    } else {
        printf("Final result is empty.\n");
    }
    
    printf("Final processed result contains %zu combinations\n", finalVectors.size());
    
    // Open file for writing
    FILE* outFile = fopen("zdd.bin", "wb");
    if (!outFile) {
        fprintf(stderr, "Error: Could not open zdd.bin for writing\n");
    } else {
        for (const auto& vec : finalVectors) {
            int size = vec.size();
            fwrite(&size, sizeof(int), 1, outFile);
            fwrite(vec.data(), sizeof(int), size, outFile);
        }
        fclose(outFile);
        printf("Results written to zdd.bin\n");
    }
    
    // Clean up original sets
    for (size_t i = 0; i < cudaSets.size(); i++) {
        freeCudaSet(&cudaSets[i]);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

//-------------------------------------------------------------------------
// Main function
//-------------------------------------------------------------------------

int main(int argc, char* argv[]) {
    // Check command line arguments
    if (argc != 2) {
        printf("Usage: %s <witness_json_file>\n", argv[0]);
        printf("Example: %s witness_export.json\n", argv[0]);
        return EXIT_FAILURE;
    }
    
    std::string filename = argv[1];
    
    // Initialize CUDA
    int deviceCount;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found.\n");
        return EXIT_FAILURE;
    }
    CHECK_CUDA_ERROR(hipSetDevice(0));
    
    printf("Processing Witness JSON file: %s\n", filename.c_str());
    
    // Run Witness test cases
    runWitnessTestCases(filename);
    
    // Clean up
    CHECK_CUDA_ERROR(hipDeviceReset());
    
    return 0;
}
